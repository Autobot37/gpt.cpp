#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
using namespace std;

hipblasHandle_t handle;

void gemm(float* out, float* in, float* w, float* b, int N, int D) {
    float alpha = 1.0;
    int lda = D;
    int incx = 1;
    float beta = 0.0;
    int incy = 1;

    hipblasStatus_t status =  hipblasSgemv(handle, HIPBLAS_OP_T, D, N, &alpha, w, lda, in, incx, &beta, out, incy);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemv failed\n");
    }
}

int main(){

    hipblasCreate(&handle);
    int N = 4096;
    int D = 4096;
    float *in, *w, *out, *b;
    hipMalloc(&in, N * D * sizeof(float));
    hipMalloc(&w, D * N * sizeof(float));
    hipMalloc(&out, N * sizeof(float));
    hipMalloc(&b, N * sizeof(float));

    gemm(out, in, w, b, N, D);

    hipblasDestroy(handle);

    return 0;
}
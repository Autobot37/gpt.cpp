#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define TILESIZE 8

__global__ void matmul_gpu_kernel(float* out, float* inp, float* weight, float* bias, int B, int T, int C, int OC){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    //since we are using 3 level parellisation we are sitting on index that is [b,t,c] 
    //so we need to do out = inp @ weight
    int b = bz * blockDim.z + tz;
    int t = bx * blockDim.x + tx;
    int t2 = by * blockDim.y + ty;

    if(b<B && t<T && t2<OC){
        float* inp_p = inp + b * T * C + t * C;
        float* out_p = out + b * T * OC + t * OC;
        float* weight_p = weight + t2* C;
        float val = (bias != NULL) ? bias[t2] : 0.0f;
        for(int i = 0;i<C;i++){
            val += inp_p[i] * weight_p[i];
        }
        out_p[t2] = val;
    }
    
}
void matmul_forward_gpu(float* out, float* inp, float* weight, float* bias, int B, int T, int C, int OC){
    dim3 block_size(8, 8, 16);
    dim3 grid_size;
    grid_size.x = (T + block_size.y - 1) / block_size.y;
    grid_size.y = (OC + block_size.z - 1) / block_size.z;
    grid_size.z = (B + block_size.x - 1) / block_size.x;
    matmul_gpu_kernel<<<grid_size, block_size>>>(out, inp, weight, bias, B, T, C, OC);
}
//inp(B,T,C) @  weight(3*C, C).T -> out(B,T,3*C)
void matmul_forward(float* out, float* inp, float* weight, float* bias, int B, int T, int C, int OC){
    #pragma omp parallel for
    for(int b = 0;b<B;b++){
        for(int t = 0;t<T;t++){
            float* inp_p = inp + b * T * C + t * C;
            float* out_p = out + b * T * OC + t * OC;
            for(int t2=0;t2<OC;t2++){
                float* weight_p = weight + t2* C;
                float val = (bias != NULL) ? bias[t2] : 0.0f;
                for(int i = 0;i<C;i++){
                    val += inp_p[i] * weight_p[i];
                }
                out_p[t2] = val;
            }
        }
    }
}

int main(){

    int B = 16;
    int T = 1024;
    int C = 768;
    int OC = C * 3;

    float *inp, *weight, *bias, *out;
    float *d_inp, *d_weight, *d_bias, *d_out;
    inp = (float*)malloc(B*T*C*sizeof(float));
    weight = (float*)malloc(OC*C*sizeof(float));
    bias = (float*)malloc(OC*sizeof(float));
    out = (float*)malloc(B*T*OC*sizeof(float));

    hipMalloc(&d_inp, B*T*C*sizeof(float));
    hipMalloc(&d_weight, OC*C*sizeof(float));
    hipMalloc(&d_bias, OC*sizeof(float));
    hipMalloc(&d_out, B*T*OC*sizeof(float));

    clock_t start, mid, end;
    double cpu_time_used, gpu_time_used;
    start = clock();
    matmul_forward(out, inp, weight, bias, B, T, C, OC);
    mid = clock();
    cpu_time_used = ((double) (mid - start)) / CLOCKS_PER_SEC;

    matmul_forward_gpu(d_out, d_inp, d_weight, d_bias, B, T, C, OC);
    hipDeviceSynchronize();
    end = clock();
    gpu_time_used = ((double) (end - mid)) / CLOCKS_PER_SEC;

    float* check;
    check = (float*)malloc(B*T*OC*sizeof(float));
    hipMemcpy(check, d_out, B*T*OC*sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0;i<B*T*OC;i++){
        if(abs(out[i] - check[i] > 1e-5)){
            printf("Incorrect output try again!\n");
            return 1;
        }
    }
    printf("CPU time used: %f\n", cpu_time_used);
    printf("GPU time used: %f\n", gpu_time_used);
    int faster = (int)(cpu_time_used / gpu_time_used);
    printf("GPU is %d times faster than CPU\n", faster);
    printf("And Correct too!\n");

    free(inp);
    free(weight);
    free(bias);
    free(out);
    free(check);
    hipFree(d_inp);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_out);





    return 0;
}
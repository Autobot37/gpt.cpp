#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

void layernorm_forward(float* out, float* mean, float* std_dev, float* inp, float* weight, float* bias, int B, int T, int C){
    float eps = 1e-5f;
    for(int b = 0;b<B;b++){
        for(int t = 0;t<T;t++){
            float* inp_p = inp + b*T*C + t*C;
            float* out_p = out + b*T*C + t*C;
                        
            float m = 0.0f;
            for(int i=0;i<C;i++){
                m += inp_p[i];
            }
            m = m/C;
            mean[b*T + t] = m;

            float v = 0.0f;
            for(int i = 0;i<C;i++){
                float diff = inp_p[i] - m;
                v += diff * diff;
            }
            v = v / C;
            float s = 1.0f/sqrtf(v + eps);
            std_dev[b*T + t] = s;

            for(int i = 0;i<C;i++){
                out_p[i] = ((inp_p[i] - m) * s) * weight[i] + bias[i];            
            }
        }    
    }
}

__global__ void layernorm_forward_kernel(float* out, float* mean, float* std_dev, float* inp, float* weight, float* bias, int B, int T, int C){
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int t = blockIdx.y * blockDim.y + threadIdx.y;
    if(b>=B || t>=T){
        return;
    }
    float eps = 1e-5f;
    float* inp_p = inp + b*T*C + t*C;
    float* out_p = out + b*T*C + t*C;
                
    float m = 0.0f;
    for(int i=0;i<C;i++){
        m += inp_p[i];
    }
    m = m/C;
    mean[b*T + t] = m;

    float v = 0.0f;
    for(int i = 0;i<C;i++){
        float diff = inp_p[i] - m;
        v += diff * diff;
    }
    v = v / C;
    float s = 1.0f/sqrtf(v + eps);
    std_dev[b*T + t] = s;

    for(int i = 0;i<C;i++){
        out_p[i] = ((inp_p[i] - m) * s) * weight[i] + bias[i];            
    }
}

void layernorm_forward_gpu(float* out, float* mean, float* std_dev, float* inp, float* weight, float* bias, int B, int T, int C){
    dim3 threadsPerBlock(4, 256);
    dim3 numBlocks((B + threadsPerBlock.x - 1)/threadsPerBlock.x, (T + threadsPerBlock.y - 1)/threadsPerBlock.y);
    layernorm_forward_kernel<<<numBlocks, threadsPerBlock>>>(out, mean, std_dev, inp, weight, bias, B, T, C);
    hipDeviceSynchronize();
}

void rand_init(float* arr, int size){
    for(int i = 0;i<size;i++){
        arr[i] = (float)rand() / RAND_MAX;
    }
}

int main(){

    int mul = 4;
    int B = 4*mul;
    int T = 128*mul;
    int C = 128*mul;

    float* inp = (float*)malloc(B*T*C*sizeof(float));
    float* out = (float*)malloc(B*T*C*sizeof(float));
    float* mean = (float*)malloc(B*T*sizeof(float));
    float* std_dev = (float*)malloc(B*T*sizeof(float));
    float* weight = (float*)malloc(C*sizeof(float));
    float* bias = (float*)malloc(C*sizeof(float));
    rand_init(inp, B*T*C);
    rand_init(weight, C);
    rand_init(bias, C);

    clock_t start, end;
    double time_used;
    start = clock();
    layernorm_forward(out, mean, std_dev, inp, weight, bias, B, T, C);
    end = clock();
    time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("CPU Time used: %f\n", time_used);

    //GPU
    float *d_inp, *d_out, *d_mean, *d_std_dev, *d_weight, *d_bias;
    hipMalloc(&d_inp, B*T*C*sizeof(float));
    hipMalloc(&d_out, B*T*C*sizeof(float));
    hipMalloc(&d_mean, B*T*sizeof(float));
    hipMalloc(&d_std_dev, B*T*sizeof(float));
    hipMalloc(&d_weight, C*sizeof(float));
    hipMalloc(&d_bias, C*sizeof(float));

    hipMemcpy(d_inp, inp, B*T*C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, C*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    start = clock();
    layernorm_forward_gpu(d_out, d_mean, d_std_dev, d_inp, d_weight, d_bias, B, T, C);
    end = clock();
    time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("GPU Time used: %f\n", time_used);

    float* check;
    check = (float*)malloc(B*T*C*sizeof(float));
    hipMemcpy(check, d_out, B*T*C*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i=0;i<B*T*C;i++){
        if(abs(out[i] - check[i]) > 1e-3f){
            printf("Incorrect Output Try Again!\n");
            return 0;
        }
    }
    printf("And its Correct too! Yay!\n");

    return 0;
}
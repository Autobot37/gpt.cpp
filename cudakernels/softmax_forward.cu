#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

void softmax_forward(float* out, float* inp, int B, int T, int V){
    for(int b=0;b<B;b++){
        for(int t=0;t<T;t++){
            float* inp_p = inp + b * T * V + t * V;
            float* out_p = out + b * T * V + t * V;
            float max_val = -1e-5f;
            for(int i=0;i<V;i++){
                if(inp_p[i] > max_val){
                    max_val = inp_p[i];
                }
            }
            float sum = 0.0f;
            for(int i=0;i<V;i++){
                out_p[i] = expf(inp_p[i] - max_val);
                sum += out_p[i];
            }
            for(int i=0;i<V;i++){
                out_p[i] = out_p[i] / sum;
            }
        }
    }
}

__global__ void softmax_forward_kernel(float* out, float* inp, int B, int T, int V){
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int t = blockIdx.y * blockDim.y + threadIdx.y;
    if(b>=B || t>=T){
        return;
    }
    float* inp_p = inp + b * T * V + t * V;
    float* out_p = out + b * T * V + t * V;
    float max_val = -1e-5f;
    for(int i=0;i<V;i++){
        if(inp_p[i] > max_val){
            max_val = inp_p[i];
        }
    }
    float sum = 0.0f;
    for(int i=0;i<V;i++){
        out_p[i] = expf(inp_p[i] - max_val);
        sum += out_p[i];
    }
    for(int i=0;i<V;i++){
        out_p[i] = out_p[i] / sum;
    }
}

void softmax_forward_gpu(float* out, float* inp, int B, int T, int V){
    dim3 threads(4,256);
    dim3 blocks((B + threads.x-1)/threads.x, (T + threads.y-1)/threads.y);
    softmax_forward_kernel<<<blocks, threads>>>(out, inp, B, T, V);
}

void rand_init(float* arr, int size){
    for(int i = 0;i<size;i++){
        arr[i] = (float)rand() / RAND_MAX;
    }
}
void print_mat(float* arr, int size){
    for(int i =0;i<size;i++){
	printf("%f",arr[i]);
    }
}

int main(){

    int mul = 1;
    int B = 16*mul;
    int T = 1024*mul;
    int V = 8192*mul;

    float* inp = (float*)malloc(B*T*V*sizeof(float));
    float* out = (float*)malloc(B*T*V*sizeof(float));
    rand_init(inp, B*T*V);
    
    clock_t start, end;
    start = clock();
    softmax_forward(out, inp, B, T, V);
    end = clock();
    printf("Time taken by CPU: %f\n", (double)(end-start)/CLOCKS_PER_SEC);

    //gpu
    float* d_inp;
    float* d_out;
    hipMalloc(&d_inp, B*T*V*sizeof(float));
    hipMalloc(&d_out, B*T*V*sizeof(float));

    hipMemcpy(d_inp, inp, B*T*V*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    start = clock();
    softmax_forward_gpu(d_out, d_inp, B, T, V);
    hipDeviceSynchronize();
    end = clock();
    printf("Time taken by GPU: %f\n", (double)(end-start)/CLOCKS_PER_SEC);

    float* check = (float*)malloc(B*T*V*sizeof(float));
    hipMemcpy(check, d_out, B*T*V*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    //print_mat(check,B*T*V);
    //print_mat(out, B*T*V);

    for(int i=0;i<B*T*V;i++){
        if(fabs(out[i] - check[i]) > 1e-5f){
            printf("Incorrect output Try Again!\n");
            return 1;
        }
    }
    printf("Correct output Yay!\n");
    
    return 0;
}

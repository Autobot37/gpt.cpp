#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

void attention_forward(float* out, float* preatt, float* att, float* qkv, int B, int T, int C, int NH){
    int hs = C/NH;
    float scale = 1.0 / sqrtf(hs);
    #pragma omp parallel for collapse(3) schedule(dynamic)
    for(int b = 0;b<B;b++){
        for(int t = 0;t<T;t++){
            for(int h = 0;h<NH;h++){
                
                //q @ k 
                float* query = qkv + b * T * 3 * C + t * 3 * C + h * hs;
                float* preatt_p = preatt + b*NH*T*T + h*T*T + t*T;
                float* att_p = att + b*NH*T*T + h*T*T + t*T;

                float maxval = 1e-5f;
                #pragma omp simd reduction(max:maxval)
                for(int t2=0;t2<=t;t2++){
                    float* key = qkv + b * T * 3 * C + t2 * 3 * C + h*hs + C;
                    float val = 0.0f;
                    #pragma omp simd reduction(+:val)
                    for(int i = 0;i<hs;i++){
                        val += query[i] * key[i];
                    }
                    val *= scale;
                    if(val>maxval){
                        maxval = val;
                    }
                    preatt_p[t2] = val;
                }
                //softmax
                float sum = 0.0f;
                #pragma omp simd reduction(+:sum)
                for(int t2=0;t2<=t;t2++){
                    float val = expf(preatt_p[t2] - maxval);
                    att_p[t2] = val;
                    sum += val;
                }
                float expinv = (sum==0.0f) ? 0.0f : 1.0f/sum;
                #pragma omp simd
                for(int t2=0;t2<T;t2++){
                    if(t2<=t){
                        att_p[t2] *= expinv;
                    }
                    else{
                        att_p[t2] = 0.0f;
                    }
                }   
                //accumulating
                float* out_p = out + b*T*C + t*C + h*hs;
                #pragma omp simd
                for(int t2=0;t2<hs;t2++){
                    float val = 0.0f;
                    #pragma omp simd reduction(+:val)
                    for(int i = 0;i<T;i++){
                        float value = qkv[b*T*3*C + i*3*C + 2*C + h*hs + t2];
                        val += att_p[i] * value;
                    }
                    out_p[t2] = val;
                }
            }
        }
    }
} 
//ok first parellizing across only B and T

__global__ void attention_forward_kernel(float* out, float* preatt, float* att, float* qkv, int B, int T, int C, int NH){
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int t = blockIdx.y * blockDim.y + threadIdx.y;
    if(b>=B || t>=T){
        return;
    }
    int hs = C/NH;
    float scale = 1.0 / sqrtf(hs);
    for(int h = 0;h<NH;h++){
        float* query = qkv + b * T * 3 * C + t * 3 * C + h * hs;
        float* preatt_p = preatt + b*NH*T*T + h*T*T + t*T;
        float* att_p = att + b*NH*T*T + h*T*T + t*T;

        float maxval = 1e-5f;
        #pragma omp simd reduction(max:maxval)
        for(int t2=0;t2<=t;t2++){
            float* key = qkv + b * T * 3 * C + t2 * 3 * C + h*hs + C;
            float val = 0.0f;
            #pragma omp simd reduction(+:val)
            for(int i = 0;i<hs;i++){
                val += query[i] * key[i];
            }
            val *= scale;
            if(val>maxval){
                maxval = val;
            }
            preatt_p[t2] = val;
        }
        //softmax
        float sum = 0.0f;
        #pragma omp simd reduction(+:sum)
        for(int t2=0;t2<=t;t2++){
            float val = expf(preatt_p[t2] - maxval);
            att_p[t2] = val;
            sum += val;
        }
        float expinv = (sum==0.0f) ? 0.0f : 1.0f/sum;
        #pragma omp simd
        for(int t2=0;t2<T;t2++){
            if(t2<=t){
                att_p[t2] *= expinv;
            }
            else{
                att_p[t2] = 0.0f;
            }
        }   
        //accumulating
        float* out_p = out + b*T*C + t*C + h*hs;
        #pragma omp simd
        for(int t2=0;t2<hs;t2++){
            float val = 0.0f;
            #pragma omp simd reduction(+:val)
            for(int i = 0;i<T;i++){
                float value = qkv[b*T*3*C + i*3*C + 2*C + h*hs + t2];
                val += att_p[i] * value;
            }
            out_p[t2] = val;
        }
    }
}
void attention_forward_gpu(float* out, float* preatt, float* att, float* qkv, int B, int T, int C, int NH){
    dim3 threads(32, 32);
    dim3 blocks((B+31)/32, (T+31)/32);
    attention_forward_kernel<<<blocks, threads>>>(out, preatt, att, qkv, B, T, C, NH);
    hipDeviceSynchronize();
}

void rand_init(float* arr, int size){
    for(int i = 0;i<size;i++){
        arr[i] = (float)rand() / RAND_MAX;
    }
}

int main(){

    int mul = 4;
    int B = 4*mul;
    int T = 128*mul;
    int C = 128*mul;
    int OC = 128*2*mul;
    int NH = 8*mul;

    float *preatt, *att, *qkv, *out;
    preatt = (float*)malloc(B*NH*T*T*sizeof(float));
    att = (float*)malloc(B*NH*T*T*sizeof(float));
    qkv = (float*)malloc(B*T*3*C*sizeof(float));
    out = (float*)malloc(B*T*C*sizeof(float));
    rand_init(preatt, B*NH*T*T);
    rand_init(att, B*NH*T*T);
    rand_init(qkv, B*T*3*C);

    srand(time(NULL));
    clock_t start, end;
    double time_used;
    start = clock();
    attention_forward(out, preatt, att, qkv, B, T, C, NH);
    end = clock();
    time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time: %f\n", time_used);

    //gpu
    float *d_preatt, *d_att, *d_qkv, *d_out;
    hipMalloc(&d_preatt, B*NH*T*T*sizeof(float));
    hipMalloc(&d_att, B*NH*T*T*sizeof(float));
    hipMalloc(&d_qkv, B*T*3*C*sizeof(float));
    hipMalloc(&d_out, B*T*C*sizeof(float));

    hipMemcpy(d_preatt, preatt, B*NH*T*T*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_att, att, B*NH*T*T*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_qkv, qkv, B*T*3*C*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    start = clock();
    attention_forward_gpu(d_out, d_preatt, d_att, d_qkv, B, T, C, NH);
    end = clock();
    time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time: %f\n", time_used);

    float* check_out;
    check_out = (float*)malloc(B*T*C*sizeof(float));
    hipMemcpy(check_out, d_out, B*T*C*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i=0;i<B*T*C;i++){
        if(abs(out[i] - check_out[i]) > 1e-3f){
            printf("Incorrect output Try Again!\n");
            return 0;
        }
    }
    printf("Correct output Yay!\n");

    return 0;
}
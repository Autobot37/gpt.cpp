#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>
using namespace std;

void matmul(float* out, float* in, float* w, float* b, int N ,int D){
    //in is D, w is N,D, b is N, out is N
    int i;
    #pragma omp parallel for private(i)
    for(i = 0;i<N;i++){
        float sum = (b!=NULL) ? b[i] : 0;
        for(int j = 0;j<D;j++){
            sum += in[j] * w[i*D + j];
        }
        out[i] = sum;
    } 
}

__global__
void matmul_kernel(float* out, float* in, float* w, float* b, int N ,int D){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        float sum = (b!=NULL) ? b[i] : 0;
        for(int j = 0;j<D;j++){
            sum += in[j] * w[i*D + j];
        }
        out[i] = sum;
    }
}

void matmul_gpu(float* out, float* in, float* w, float* b, int N ,int D){
    int num_threads = 1024;
    int num_blocks = (N + num_threads - 1) / num_threads;
    matmul_kernel<<<num_blocks, num_threads>>>(out, in, w, b, N, D);
}

__global__ void add_bias(float* out, float* b, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        out[i] += b[i];
    }
}

void gemm(float* out, float* in, float* w, float* b, int N, int D) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0;
    int lda = D;
    int incx = 1;
    float beta = 0.0;
    int incy = 1;

    hipblasStatus_t status =  hipblasSgemv(handle, HIPBLAS_OP_T, D, N, &alpha, w, lda, in, incx, &beta, out, incy);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("hipblasSgemv failed\n");
    }
    if(b != NULL)
    add_bias<<<(N + 1023) / 1024, 1024>>>(out, b, N);

    hipblasDestroy(handle);
}

void rand_init(float* arr, int N){
    for(int i = 0;i<N;i++){
        arr[i] = (float)rand() / RAND_MAX;
    }
}

void isequal(float* a, float* b, int n){
    float maxval = -INFINITY;
    for(int i = 0;i<n;i++){
        maxval = fmaxf(maxval, fmaxf(a[i], b[i]));
    }
    float eps = 1e-5;
    for(int i = 0;i<n;i++){
        if(fabs(a[i] - b[i]) > eps * (maxval + 1)){
            cout << "Mismatch at index " << i << " CPU: " << a[i] << " GPU: " << b[i] << endl;
        }
    }
}

int main(){

    int N = 1024;
    int D = 2048;

    float* in = (float*)malloc(D * sizeof(float));
    float* w = (float*)malloc(N * D * sizeof(float));
    float* b = (float*)malloc(N * sizeof(float));
    float* out = (float*)malloc(N * sizeof(float));

    rand_init(in, D);
    rand_init(w, N * D);
    rand_init(b, N);

    float* d_in, *d_w, *d_b, *d_out;
    hipMalloc(&d_in, D * sizeof(float));
    hipMalloc(&d_w, N * D * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));

    hipMemcpy(d_in, in, D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, N * D * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    matmul(out, in, w, b, N, D);

    matmul_gpu(d_out, d_in, d_w, d_b, N, D);

    float* out_gpu = (float*)malloc(N * sizeof(float));
    hipMemcpy(out_gpu, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    isequal(out, out_gpu, N);

    //----sgemm
    float* out_gemm;
    hipMalloc(&out_gemm, N * sizeof(float));
    gemm(out_gemm, d_in, d_w, d_b, N, D);

    float* out_gemm_cpu = (float*)malloc(N * sizeof(float));
    hipMemcpy(out_gemm_cpu, out_gemm, N * sizeof(float), hipMemcpyDeviceToHost);

    isequal(out, out_gemm_cpu, N);

    return 0;
}
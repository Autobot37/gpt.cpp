#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <omp.h>

__global__ void matmul_gpu_kernel(float* out, float* inp, float* weight, float* bias, int B, int T, int C, int OC){
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int bt = bx * blockDim.x + tx;
    int oc = by * blockDim.y + ty;

    if(bt<B*T && oc<OC){
        float* inp_p = inp + bt * C;
        float* weight_p = weight + oc * C;

        float val = (bias != NULL) ? bias[oc] : 0.0f;
        for(int i = 0;i<C;i++){
            val += inp_p[i] * weight_p[i];
        }
        out[bt * OC + oc] = val;
    }
}
void matmul_forward_gpu(float* out, float* inp, float* weight, float* bias, int B, int T, int C, int OC){
    dim3 block_size(32, 32);
    dim3 grid_size;
    grid_size.x = (B*T + block_size.x - 1) / block_size.x;
    grid_size.y = (OC + block_size.y - 1) / block_size.y;
    matmul_gpu_kernel<<<grid_size, block_size>>>(out, inp, weight, bias, B, T, C, OC);
}
//inp(B,T,C) @  weight(3*C, C).T -> out(B,T,3*C)
void matmul_forward(float* out, float* inp, float* weight, float* bias, int B, int T, int C, int OC){
    #pragma omp parallel for collapse(2) schedule(static)    
    for(int b = 0;b<B;b++){
        for(int t = 0;t<T;t++){
            float* inp_p = inp + b * T * C + t * C;
            float* out_p = out + b * T * OC + t * OC;
            for(int o=0;o<OC;o++){
                float* weight_p = weight + o * C;
                float val = (bias != NULL) ? bias[o] : 0.0f;
                #pragma omp simd reduction(+:val) 
                for(int i = 0;i<C;i++){
                    val += inp_p[i] * weight_p[i];
                }
                out_p[o] = val;
            }
        }
    }
}

void rand_init(float* arr, int size){
    for(int i = 0;i<size;i++){
        arr[i] = (float)rand() / RAND_MAX;
    }
}
void print_3d(float* arr, int B, int T, int C){
    for(int b = 0;b<B;b++){
        for(int t = 0;t<T;t++){
            for(int c = 0;c<C;c++){
                printf("%f ", arr[b*T*C + t*C + c]);
            }
            printf("\n");
        }
        printf("\n");
    }
}

int main(){

    int mul = 1;
    int B = 1*mul;
    int T = 4*mul;
    int C = 4*mul;
    int OC = 4*mul;

    float *inp, *weight, *bias, *out;
    float *d_inp, *d_weight, *d_bias, *d_out;
    inp = (float*)malloc(B*T*C*sizeof(float));
    weight = (float*)malloc(OC*C*sizeof(float));
    bias = (float*)malloc(OC*sizeof(float));
    out = (float*)malloc(B*T*OC*sizeof(float));
    rand_init(inp, B*T*C);
    rand_init(weight, OC*C);
    rand_init(bias, OC);
    float* check;
    check = (float*)malloc(B*T*OC*sizeof(float));

    hipMalloc(&d_inp, B*T*C*sizeof(float));
    hipMalloc(&d_weight, OC*C*sizeof(float));
    hipMalloc(&d_bias, OC*sizeof(float));
    hipMalloc(&d_out, B*T*OC*sizeof(float));

    hipMemset(d_inp, 0, B*T*C*sizeof(float));
    hipMemset(d_weight, 0, OC*C*sizeof(float));
    hipMemset(d_bias, 0, OC*sizeof(float));
    hipMemset(d_out, 0, B*T*OC*sizeof(float));

    hipMemcpy(d_inp, inp, B*T*C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, weight, OC*C*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, OC*sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    float* check_input;
    check_input = (float*)malloc(B*T*C*sizeof(float));
    hipMemcpy(check_input, d_inp, B*T*C*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    print_3d(inp, B, T, C);
    print_3d(check_input, B, T, C);

    for(int i = 0;i<B*T*C;i++){
        if(abs(inp[i] - check_input[i]) > 1e-3f){
            printf("Incorrect even input man Try again!\n");
            return 0;
        }
    }

    clock_t start, mid, end;
    double cpu_time_used, gpu_time_used;
    start = clock();
    matmul_forward(out, inp, weight, bias, B, T, C, OC);
    mid = clock();
    cpu_time_used = ((double) (mid - start)) / CLOCKS_PER_SEC;

    matmul_forward_gpu(d_out, d_inp, d_weight, d_bias, B, T, C, OC);
    hipDeviceSynchronize();
    end = clock();
    gpu_time_used = ((double) (end - mid)) / CLOCKS_PER_SEC;

    hipMemcpy(check, d_out, B*T*OC*sizeof(float), hipMemcpyDeviceToHost);

    printf("CPU time used: %f\n", cpu_time_used);
    printf("GPU time used: %f\n", gpu_time_used);
    int faster = (int)(cpu_time_used / gpu_time_used);
    printf("GPU is %d times faster than CPU\n", faster);

    print_3d(out, B, T, OC);
    print_3d(check, B, T, OC);

    for(int i = 0;i<B*T*OC;i++){
        if(abs(out[i] - check[i] > 1e-3f)){
            printf("Incorrect output Try again!\n");
            return 0;
        }
    }

    printf("And Correct too!\n");

    free(inp);
    free(weight);
    free(bias);
    free(out);
    free(check);
    hipFree(d_inp);
    hipFree(d_weight);
    hipFree(d_bias);
    hipFree(d_out);





    return 0;
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "../utils.h"

void encoder_forward(float* out, int* inp, float* wte, float* wpe, int B, int T, int C){
    for(int b = 0;b<B;b++){
        for(int t = 0;t<T;t++){
            float* out_p = out + b * T * C + t * C;
            float* wte_p = wte + inp[b * T + t] * C;
            float* wpe_p = wpe + t * C;
            for(int i = 0;i<C;i++){
                out_p[i] = wte_p[i] + wpe_p[i];
            }
        }
    };
}

__global__ void encoder_forward_kernel(float* out, int* inp, float* wte, float* wpe, int B, int T, int C){
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    int t = blockIdx.y * blockDim.y + threadIdx.y;

    if (b < B && t < T) {
        float* out_p = out + b * T * C + t * C;
        float* wte_p = wte + inp[b * T + t] * C;
        float* wpe_p = wpe + t * C;
        for(int i = 0; i < C; i++) {
            out_p[i] = wte_p[i] + wpe_p[i];
        }
    }
}

void encoder_forward_gpu(float* out, int* inp, float* wte, float* wpe, int B, int T, int C){
    dim3 blockDim(32, 32); 
    dim3 gridDim((B + blockDim.x - 1) / blockDim.x, (T + blockDim.y - 1) / blockDim.y); // Adjust grid size
    encoder_forward_kernel<<<gridDim, blockDim>>>(out, inp, wte, wpe, B, T, C);
    hipDeviceSynchronize();
    
}

void rand_init(float* arr, int size){
    for(int i = 0;i<size;i++){
        arr[i] = (float)rand() / RAND_MAX;
    }
}

int main(){

    int mul = 8;
    int C = 32*mul;
    int NH = 4*mul;
    int T = 128*mul;
    int V = 4096*mul;
    int L = 4*mul;
    int B = 1*mul;
    int* inp = (int*)mallocCheck(sizeof(int) * B * T);
    float* out = (float*)mallocCheck(sizeof(int) * B * T * C);
    float* wpe = (float*)mallocCheck(sizeof(int) * T * C);
    float* wte = (float*)mallocCheck(sizeof(int) * V * C);
    rand_init(wpe, T * C);
    rand_init(wte, V * C);
    for(int i = 0;i<B*T;i++){
        inp[i] = rand() % V;
    }

    clock_t start, end, end2;
    double time_used;
    start = clock();

    encoder_forward(out, inp, wte, wpe, B,T,C);

    end = clock();
    time_used = ((double)(end - start)) / CLOCKS_PER_SEC;  
    printf("Time Used CPU: %lf seconds\n", time_used);
    
    //
    float *d_out;
    int *d_inp;
    float *d_wte;
    float *d_wpe;
    hipMalloc(&d_out, B * T * C * sizeof(float));
    hipMalloc(&d_inp, B * T * sizeof(int));
    hipMalloc(&d_wte, V * C * sizeof(float)); 
    hipMalloc(&d_wpe, T * C * sizeof(float));
    hipMemcpy(d_inp, inp, B * T * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_wte, wte, V * C * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_wpe, wpe, T * C * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    encoder_forward_gpu(out, inp, wte, wpe, B,T,C);

    end2 = clock();
    time_used = ((double)(end2 - end)) / CLOCKS_PER_SEC;
    printf("Time Used GPU: %lf seconds\n", time_used);

    float* check;
    check = (float*)mallocCheck(sizeof(float) * B * T * C);
    hipMemcpy(check, d_out, B * T * C * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for(int i = 0;i<B*T*C;i++){
        if(abs(out[i] - check[i]) > 1e-3f){
            printf("Incorrect output Try Again\n");
            return 1;
        }
    }
    printf("Correct output Yay!\n");
            
    free(inp);
    free(out);
    free(wpe);
    free(wte);
    hipFree(d_inp);
    hipFree(d_out);
    hipFree(d_wpe);
    hipFree(d_wte);


    return 0;
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

//n,m @ m,k -> n k
void matmul_cpu(float* out, float* a, float* b, int N, int M, int K){
    #pragma omp parallel for collapse(2)
    for(int i=0;i<N;i++){
        for(int j=0;j<K;j++){
            float sum = 0;
            #pragma omp parallel for reduction(+:sum)
            for(int k = 0;k<M;k++){
                sum += a[i*M+k] * b[k*K+j];
            }
            out[i*K + j] = sum;
        }
    }
}

#define TILESIZE 32

__global__ void matmul_gpu_kernel(float* out, float* a, float* b, int N, int M, int K){
    __shared__ int A[TILESIZE][TILESIZE];
    __shared__ int B[TILESIZE][TILESIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float sum = 0.0f;
    for(int i = 0;i< M / TILESIZE;i++){
        A[ty][tx] = a[row * M + i * TILESIZE + tx];
        B[ty][tx] = b[(i * TILESIZE + ty) * K + col];
        __syncthreads();

        for(int j=0;j<TILESIZE;j++){
            sum += A[ty][j] * B[j][tx];
        }
        __syncthreads();
    }
    out[row * K + col] = sum;
}
void matmul_gpu(float* out, float* a, float* b, int N, int M, int K){
    dim3 block_size(32, 32);
    dim3 grid_size;
    grid_size.x = (K + block_size.x - 1) / block_size.x;
    grid_size.y = (N + block_size.y - 1) / block_size.y;
    matmul_gpu_kernel<<<grid_size, block_size>>>(out, a, b, N, M, K);
}

int main(){ 
    
    int N = 1024;
    int M = 1024*4;
    int K = 1024;
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    a = (float*)malloc(N*M*sizeof(float));
    b = (float*)malloc(M*K*sizeof(float));
    out = (float*)malloc(N*K*sizeof(float));
    hipMalloc(&d_a, N*M*sizeof(float));
    hipMalloc(&d_b, M*K*sizeof(float));
    hipMalloc(&d_out, N*K*sizeof(float));

    hipMemcpy(d_a, a, N*M*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, M*K*sizeof(float), hipMemcpyHostToDevice);


    clock_t start_time = clock();
    matmul_gpu(out, a, b, N, M, K);
    clock_t mid_time = clock();
    double time1 = (double)(mid_time - start_time) / CLOCKS_PER_SEC;
    printf("GPU time: %f\n", time1);

    matmul_cpu(out, a, b, N, M, K);
    double time2 = (double)(clock() - mid_time) / CLOCKS_PER_SEC;
    printf("CPU time: %f\n", time2);

    float* check = (float*)malloc(N*K*sizeof(float));
    hipMemcpy(check, d_out, N*K*sizeof(float), hipMemcpyDeviceToHost);
    for(int i=0;i<N*K;i++){
        if((out[i] - check[i]) > 1e-5){
            printf("Error at %d\n", i);
            break;
        }
    }
    int faster = (int)time2 / time1;
    printf("GPU is %d times faster than CPU\n", faster);
    printf("And it is correct too\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
    free(check);
    
    return 0;
}